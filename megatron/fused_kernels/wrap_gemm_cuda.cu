#include "hip/hip_runtime.h"

#include "wrap_gemm_cuda.hpp"

#include <iostream>
#include <stdexcept>

#include <hip/hip_runtime.h>
#include <hipblas.h>


namespace wrap_gemm {

// C += A * B
void wrap_gemm_bf16bf16bf16_f32_nn_beta1_cuda(intptr_t A_intptr, intptr_t B_intptr, intptr_t C_intptr, int m, int n, int k, intptr_t handle_intptr) {
    __hip_bfloat16 const *A = reinterpret_cast<__hip_bfloat16 const *>(A_intptr);
    __hip_bfloat16 const *B = reinterpret_cast<__hip_bfloat16 const *>(B_intptr);
    __hip_bfloat16 *C = reinterpret_cast<__hip_bfloat16 *>(C_intptr);
    hipblasHandle_t handle = reinterpret_cast<hipblasHandle_t>(handle_intptr);

    float alpha = 1.;
    float beta = 1.;

    hipblasStatus_t status = hipblasGemmEx(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        n,
        m,
        k,
        &alpha,
        B,
        HIP_R_16BF,
        n,
        A,
        HIP_R_16BF,
        k,
        &beta,
        C,
        HIP_R_16BF,
        n,
        HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS error " << (int)status << ": " << cublasGetStatusString(status) << " " << __FILE__ << ":" << __LINE__ << std::endl;
        throw std::runtime_error("cuBLAS error " + std::to_string(status) + ": " + cublasGetStatusString(status));
    }
}

void wrap_cuda_memcpy_2d_async(intptr_t dst_intptr, size_t dpitch, intptr_t src_intptr, size_t spitch, size_t width, size_t height, int cuda_memcpy_kind, intptr_t stream_intptr) {
    void *dst = reinterpret_cast<void *>(dst_intptr);
    void const *src = reinterpret_cast<void const *>(src_intptr);
    hipStream_t stream = reinterpret_cast<hipStream_t>(stream_intptr);
    hipError_t err = hipMemcpy2DAsync(dst, dpitch, src, spitch, width, height, (hipMemcpyKind)cuda_memcpy_kind, stream);
    if (err != hipSuccess) {
        std::cerr << "CUDA error " << (int)err << ": " << hipGetErrorString(err) << " " << __FILE__ << ":" << __LINE__ << std::endl;
        throw std::runtime_error("CUDA error " + std::to_string(err) + ": " + hipGetErrorString(err));
    }
}

}
